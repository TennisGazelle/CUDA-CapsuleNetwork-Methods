//
// Created by daniellopez on 4/4/18.
//

#include <iostream>
#include "CUDAUtils.h"

using namespace std;

void CUDAUtils::handleError(hipError_t error) {
    if (error != hipSuccess) {
        cerr << "CUDA error! - " << hipGetErrorString(error) << endl;
        exit(1);
    }
}