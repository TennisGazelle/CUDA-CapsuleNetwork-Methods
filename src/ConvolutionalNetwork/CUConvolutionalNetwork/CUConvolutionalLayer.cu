#include "hip/hip_runtime.h"
//
// Created by daniellopez on 4/23/18.
//

#include <cassert>
#include <CapsNetConfig.h>
#include <CUDAUtils.h>
#include <iostream>
#include "ConvolutionalNetwork/CUConvolutionalNetwork/CUConvolutionalLayer.h"

CUConvolutionalLayer::CUConvolutionalLayer(const CapsNetConfig& incomingConfig, int iHeight, int iWidth, int nFilters, int fHeight, int fWidth)
        : config(incomingConfig) {
    numFilters = nFilters;
    filterDepth = 1;
    filterHeight = fHeight;
    filterWidth = fWidth;

    inputWidth = iWidth;
    inputHeight = iHeight;

    outputHeight = inputHeight - filterHeight;
    outputWidth = inputWidth - filterWidth;

    input.resize(inputHeight*inputWidth*filterDepth);
    delta_input.resize(inputHeight*inputWidth*filterDepth);
    
    filter.resize(numFilters*filterDepth*filterHeight*filterWidth);
    filter_error.resize(numFilters*filterDepth*filterHeight*filterWidth);
    filter_velocities.resize(numFilters*filterDepth*filterHeight*filterWidth);

    output.resize(outputHeight*outputWidth*numFilters);

    filter.fillWithRandom();

    totalMemoryUsage += 2*(inputHeight*inputWidth*filterDepth);
    totalMemoryUsage += 3*(numFilters*filterDepth*filterHeight*filterWidth);
    totalMemoryUsage += outputHeight*outputWidth*numFilters;
}

void CUConvolutionalLayer::setInput(const std::vector<double>& inputImage) {
    assert(inputImage.size() == input.getSize());
    for (int i = 0; i < input.getSize(); i++) {
        input.setValueAt_1D(i, inputImage[i]/256.0);
    }
}

void CUConvolutionalLayer::setInput(const Image &inputImage) {
    assert(inputImage.size() == input.getSize());
    for (int i = 0; i < input.getSize(); i++) {
        input.setValueAt_1D(i, double(inputImage[i])/256.0);
    }
}

void CUConvolutionalLayer::forwardPropagate() {
//    input.print("input as a feature map", inputWidth);
    hipDeviceSynchronize();
    CUUnifiedBlob::CUDA_convolutionalDotProduct(input, filter, output, inputHeight, inputWidth, filterHeight, filterWidth, filterDepth, numFilters);
    if (output.CUDA_hasNan()) {
        cerr << "convolutional output has a nan: " << output.hasNan() << endl;
        output.print("output", outputWidth);
        filter.print("filter", filterWidth);
        exit(1);
     }
//    if (output.isAllZeros()) {
//        cerr << "convolutional output is all zeros..." << endl;
//        CUUnifiedBlob::CUDA_convolutionalDotProduct(input, filter, output, inputHeight, inputWidth, filterHeight, filterWidth, filterDepth, numFilters);
//        printInput();
//        printFilter();
//        printOutput();
//        exit(1);
//    }
}

void CUConvolutionalLayer::squashAndRemapToU(CUUnifiedBlob &u) {
//    output.print("conv. layer output", outputWidth);
    u.CUDA_clear();
    CUUnifiedBlob::CUDA_tensorFlatteningAndActivatedRemapping(u, output, outputHeight, outputWidth, config.cnNumTensorChannels, config.numClasses, config.cnInnerDim);
//    output.CUDA_clear();
}

void CUConvolutionalLayer::remapErrorToOutput(CUUnifiedBlob &delta_u) {
    CUUnifiedBlob::CUDA_reconstructingTensorFromError(output, delta_u, outputHeight, outputWidth, config.cnNumTensorChannels, config.numClasses, config.cnInnerDim);
//    delta_u.print("delta_u", config.cnInnerDim);
//    output.print("cvlayer error output", outputWidth);
}

void CUConvolutionalLayer::backPropagate() {
    CUUnifiedBlob::CUDA_convolutionalBackPropFromError(output, filter, filter_error, input, delta_input, inputHeight, inputWidth, filterHeight, filterWidth, filterDepth, numFilters);
//    filter.print("filter", filterWidth);
//    filter_error.print("delta filter", filterWidth);
//    if (filter.hasNan() != -1) {
//        cerr << "filter in conv. layer has nan: " << filter.hasNan() << endl;
//        cerr << "dumping..." << endl;
//        output.print("cvlayer error output", outputWidth);
//        exit(1);
//    }
//    if (filter_error.hasNan() != -1) {
//        cerr << "filter_error in conv. layer has nan: " << filter_error.hasNan() << endl;
//        cerr << "dumping..." << std::endl;
//        output.print("cvlayer error output", outputWidth);
//        exit(1);
//    }
}

void CUConvolutionalLayer::updateError() {
    CUUnifiedBlob::CUDA_elementWiseErrorUpdate(filter, filter_error, filter_velocities, filter.getSize());
}

void CUConvolutionalLayer::printFilter() const {
    filter.print("filter", filterWidth);
}

void CUConvolutionalLayer::printInput() const {
    input.print("original image input", inputWidth);
}

void CUConvolutionalLayer::printOutput() const {
    output.print("output", outputWidth);
}

int CUConvolutionalLayer::getTotalMemoryUsage() const {
    return totalMemoryUsage;
}