#include "hip/hip_runtime.h"

#include <MultilayerPerceptron/MultilayerPerceptron.h>
#include <ConvolutionalNetwork/ConvolutionalNetwork.h>
#include <ConvolutionalNetwork/ConvolutionalLayer.h>
#include <iostream>
#include <cmath>
#include <ProgressBar.h>
#include <CapsuleNetwork/Capsule.h>
#include <Utils.h>
#include <models/VectorMap.h>
#include <cassert>
#include <CapsuleNetwork/CapsuleNetwork.h>
#include <models/CUUnifiedBlob.h>

void test_SingleLayerCNN() {
    auto image = MNISTReader::getInstance()->trainingData[0];
    ConvolutionalLayer layer(28, 28, 256, 9, 9);
    MultilayerPerceptron mp(layer.getOutputSize1D(), 10, {});

    mp.init();
    layer.setInput({image.toFeatureMap()});

    layer.calculateOutput();
    vector<double> mlpOutput = mp.loadInputAndGetOutput(layer.getOutputAsOneDimensional());
    vector<double> error(mlpOutput.size());

    ProgressBar pb(5000);
    for (int iter = 0; iter < 5000; iter++) {
        for (int i = 0; i < mlpOutput.size(); i++) {
            double target = 0;
            if (i == image.getLabel()) {
                target = 1;
            };
            error[i] = mlpOutput[i] * (1-mlpOutput[i]) * (target - mlpOutput[i]);
        }
        vector<double> mlpLastLayerError = mp.backPropagateError(error);
        layer.backPropagate(
                FeatureMap::toFeatureMaps(
                        layer.outputHeight,
                        layer.outputWidth,
                        mlpLastLayerError
                )
        );

        layer.calculateOutput();
        mlpOutput = mp.loadInputAndGetOutput(layer.getOutputAsOneDimensional());

        pb.updateProgress(iter);
    }


    cout << endl;

    for (int i = 0; i < mlpOutput.size(); i++) {
        double target = 0;
        if (i == image.getLabel()) {
            target = 1;
        };
        error[i] = mlpOutput[i] * (1-mlpOutput[i]) * (target - mlpOutput[i]);
        cout << i << ": " << mlpOutput[i] << " " << error[i] << endl;
    }

    layer.printKernel(1);
    layer.printOutput(1);
}

void test_CapsuleNetSquishing() {
    int dim = 3;
    arma::vec testInput(dim, arma::fill::randn);

    testInput.print("test input...");
    Utils::squish(testInput).print("output is....");
}

void fillFeatureMapWithRandom(FeatureMap& featureMap) {
    for (auto& row : featureMap) {
        for (auto& col : row) {
            col = Utils::getWeightRand(10) + 10;
        }
    }
}


void test_VectorMapFromFeatureMaps() {
    vector<FeatureMap> inputs;
    size_t inputsDepth = 256, outputVectorDim = 8, outputsDepth = 32;
    size_t row = 6, col = 6;

    // create and fill inputs with garbage
    for (int i = 0; i < inputsDepth; i++) {
        FeatureMap fm;
        fm.setSize(row, col);
        fillFeatureMapWithRandom(fm);
        inputs.push_back(fm);
    }

    vector<VectorMap> vectorMaps = VectorMap::toSquishedVectorMap(outputVectorDim, inputs);
    assert (vectorMaps.size() == outputsDepth);

    // just check the first vector
    arma::vec singleVector = vectorMaps[0][0][0];
    arma::vec originalVector(outputVectorDim);
    for (int i = 0; i < outputVectorDim; i++) {
        originalVector[i] = inputs[i][0][0];
    }

    originalVector = Utils::squish(originalVector);
    for (int i = 0; i < outputVectorDim; i++) {
        assert (singleVector[i] == originalVector[i]);
    }
}

void test_FeatureMapsFromVectorMap() {
    size_t inputsDepth = 32, vectorDim = 8, outputsDepth = 256;
    size_t row = 6, col = 6;
    vector<arma::vec> inputs(row*col*inputsDepth, arma::vec(vectorDim, arma::fill::randu));

    for (auto& v : inputs) {
        for (auto& val : v) {
            val = Utils::getWeightRand(10) + 10;
        }
    }

    vector<FeatureMap> maps = VectorMap::toArrayOfFeatureMaps(row, col, inputsDepth, inputs);
}

void test_CapsuleNetwork_ForwardPropagation() {
    CapsuleNetwork capsuleNetwork;
    vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(0);

    for (int i = 0; i < 10; i++) {
        cout << "length of vector corresponding to " << i << ": " << sqrt(Utils::square_length(output[i])) << endl;
    }
}

void test_CapsuleNetwork_BackPropagation() {
    CapsuleNetwork capsuleNetwork;
    vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(0);
    vector<arma::vec> error = capsuleNetwork.getErrorGradient(output, MNISTReader::getInstance()->trainingData[0].getLabel());
    capsuleNetwork.backPropagate(error);
    output = capsuleNetwork.loadImageAndGetOutput(0);

    for (int i = 0; i < 10; i++) {
        cout << "length of vector corresponding to " << i << ": " << sqrt(Utils::square_length(output[i])) << endl;
    }
}

void test_CapsuleNetwork_Epoch() {
    CapsuleNetwork capsuleNetwork;

    auto& data = MNISTReader::getInstance()->trainingData;
    const size_t batchSize = 250;

    ProgressBar pb(data.size());
    for (size_t i = 0; i < data.size(); i++) {
        vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(i);
        vector<arma::vec> error = capsuleNetwork.getErrorGradient(output, data[i].getLabel());
        capsuleNetwork.backPropagate(error);

        if (i%batchSize == batchSize-1) {
            capsuleNetwork.updateWeights();
            capsuleNetwork.loadImageAndPrintOutput(i);
        }
        pb.updateProgress(i);
    }
}

void test_CapsuleNetwork_getMarginLoss() {
    CapsuleNetwork capsuleNetwork;
    vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(0);
    double totalLoss = capsuleNetwork.getTotalMarginLoss(MNISTReader::getInstance()->trainingData[0].getLabel(), output);

    cout << "total loss is: " << totalLoss << endl;
}

void test_NetworkTallyingTiming() {
    MultilayerPerceptron mp(784, 10, {16,16});
    ConvolutionalNetwork cnn;
    CapsuleNetwork capsNet;

//    mp.init();
//    cnn.init();

//    mp.runEpoch();
//    cnn.runEpoch();
//    capsNet.runEpoch();

//    mp.tally(true);
//    cnn.tally(true);
//    capsNet.tally(true); // true for training set, false for testing set

//    mp.train();
//    cnn.train();
    capsNet.train();
}

void test_CapsuleNetwork_reconstruction() {
    CapsuleNetwork capsuleNetwork;
    int targetLabel = (int) MNISTReader::getInstance()->trainingData[0].getLabel();

    vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(0);
    for (int i = 0; i < 10; i++) {
        cout << "length of vector corresponding to " << i << ": " << sqrt(Utils::square_length(output[i])) << endl;
    }
    cout << endl;

    vector<arma::vec> capsuleError = capsuleNetwork.getErrorGradient(output, targetLabel);
    vector<arma::vec> mlpError = capsuleNetwork.getReconstructionError(output, 0);

    capsuleNetwork.backPropagate(capsuleError);
    capsuleNetwork.backPropagate(mlpError);
    capsuleNetwork.updateWeights();

    vector<arma::vec> updatedOutput = capsuleNetwork.loadImageAndGetOutput(0);

    for (int i = 0; i < 10; i++) {
        cout << "length of vector corresponding to " << i << ": " << sqrt(Utils::square_length(updatedOutput[i])) << endl;
    }
}

void test_CapsuleNetwork_multipleReconstruction() {
    CapsuleNetwork capsuleNetwork;
    for (int i = 0; i < 10; i++) {
        int targetLabel = (int) MNISTReader::getInstance()->trainingData[0].getLabel();

        vector<arma::vec> output = capsuleNetwork.loadImageAndGetOutput(0);

        vector<arma::vec> capsuleError = capsuleNetwork.getErrorGradient(output, targetLabel);
        vector<arma::vec> mlpError = capsuleNetwork.getReconstructionError(output, 0);

        capsuleNetwork.backPropagate(capsuleError);
        capsuleNetwork.backPropagate(mlpError);
        capsuleNetwork.updateWeights();
    }

    vector<arma::vec> updatedOutput = capsuleNetwork.loadImageAndGetOutput(4);
    cout << "target label: " << MNISTReader::getInstance()->trainingData[4].getLabel() << endl;
    for (int i = 0; i < 10; i++) {
        cout << "length of vector corresponding to " << i << ": " << sqrt(Utils::square_length(updatedOutput[i])) << endl;
    }
}

void test_CUUnifiedBlob_CUDA_matrixVectorMultiplication() {
    int inputDim = 8, outputDim = 16, numMultiples=2;
    CUUnifiedBlob v(inputDim * numMultiples),
                  w(inputDim * outputDim * numMultiples),
                 vv(outputDim * numMultiples);

    for (int i = 0; i < inputDim; i++) {
        v.setValueAt_1D(i, i);
        v.setValueAt_1D(i+inputDim, i*10);

        w.setValueAt_2D(i, i, inputDim, 1.0);
        w.setValueAt_2D(i+inputDim, i, inputDim, 1.0);
        w.setValueAt_2D(i+2*inputDim, i, inputDim, 1.0);
        w.setValueAt_2D(i+3*inputDim, i, inputDim, 1.0);
    }
    vv.clear();

//    CUUnifiedBlob::matrixVectorMultiplication(w, v, vv, inputDim, outputDim);
    CUUnifiedBlob::CUDA_matrixVectorMultiplication(w, v, vv, inputDim, outputDim, numMultiples);

    v.print("v");
    w.print("w", inputDim);
    vv.print("vv");
}

void test_CUUnifiedBlob_CUDA_softmax() {
    int numClasses = 10, flattenedTensorSize = 45;
    CUUnifiedBlob bMatrix(numClasses * flattenedTensorSize),
                  cMatrix(numClasses * flattenedTensorSize);

    for (int k = 0; k < numClasses; k++) {
        for (int t = 0; t < flattenedTensorSize; t++) {
            bMatrix.setValueAt_2D(t, k, numClasses, t);
        }
    }
    cMatrix.clear();
    bMatrix.print("b:", numClasses);

//    CUUnifiedBlob::vectorVectorSoftmax(bMatrix, cMatrix, numClasses, flattenedTensorSize);
//    cMatrix.print("c sequentially :", numClasses);
//    cMatrix.clear();
    CUUnifiedBlob::CUDA_vectorVectorSoftmax(bMatrix, cMatrix, numClasses, flattenedTensorSize);
    cMatrix.print("c in cuda      :", numClasses);
}

void test_CUUnifiedBlob_CUDA_weightReduceAndSquash() {
    int numClasses = 2, flattenedTensorSize = 10, outputDim = 3;
    CUUnifiedBlob cMatrix(numClasses * flattenedTensorSize),
                  u_hat(numClasses * flattenedTensorSize * outputDim),
                  u_hat_cuda_output(numClasses * flattenedTensorSize * outputDim),
                  v(numClasses * outputDim),
                  v_cuda_output(numClasses * outputDim);

    int i = 1;
    for (int t = 0; t < flattenedTensorSize; t++) {
        for (int k = 0; k < numClasses; k++)  {
            cMatrix.setValueAt_2D(t, k, numClasses, i);
            for (int j = 0; j < outputDim; j++) {
                u_hat.setValueAt_2D(t, k*outputDim + j, numClasses*outputDim, i+j);
                u_hat_cuda_output.setValueAt_2D(t, k*outputDim + j, numClasses*outputDim, i+j);
            }
            i++;
        }
    }
    cMatrix.print("c", numClasses);
    u_hat.print("u_hat (original)", numClasses*outputDim);
    CUUnifiedBlob::weightReduceVectors(u_hat, cMatrix, v, numClasses, flattenedTensorSize, outputDim);
    CUUnifiedBlob::CUDA_weightReduceVectors(u_hat_cuda_output, cMatrix, v_cuda_output, numClasses, flattenedTensorSize, outputDim);

    u_hat.print("u_hat", numClasses*outputDim);
    u_hat_cuda_output.print("u_hat (cuda)", numClasses*outputDim);
    
    v.print("v", numClasses*outputDim);
    v_cuda_output.print("v (cuda)", numClasses*outputDim);

    assert(u_hat == u_hat_cuda_output);
    assert(v == v_cuda_output);
}

void test_CUUnifiedBlob_CUDA_vectorSquash() {
    int vectorDim = 8, numVectors = 2000;
	CUUnifiedBlob vectors(vectorDim * numVectors), cuda_output(vectorDim * numVectors);
	int i = 0;
	for (int v = 0; v < numVectors; v++) {
		for (int d = 0; d < vectorDim; d++) {
			vectors.setValueAt_2D(v, d, vectorDim, i);
            cuda_output.setValueAt_2D(v, d, vectorDim, i);
            i++;
		}
	}

	cuda_output.print("original (cuda)", vectorDim);
	CUUnifiedBlob::vectorSquash(vectors, numVectors, vectorDim);
    CUUnifiedBlob::CUDA_vectorSquash(cuda_output, numVectors, vectorDim);
    sleep(1);
    vectors.print("vecs", vectorDim);
    cuda_output.print("cuda output", vectorDim);
    
    assert(vectors == cuda_output);
}

void test_CUUnifiedBlob_CUDA_getScalarProducts() {
    int numClasses = 2, flattenedTensorSize = 10, dim = 3;
    CUUnifiedBlob b(numClasses * flattenedTensorSize),
                      b_cuda_output(numClasses * flattenedTensorSize),
                      u_hat(numClasses * flattenedTensorSize * dim),
                      v(numClasses * dim);

    int i = 1;
    for (int t = 0; t < flattenedTensorSize; t++) {
        for (int k = 0; k < numClasses; k++) {
    		for (int d = 0; d < dim; d++) {
    			u_hat.setValueAt_2D(t, k*dim+d, numClasses*dim, i+d);
    			v.setValueAt_2D(0, k*dim+d, numClasses*dim, i-10);
    		}
    		i++;
    	}
    }

    u_hat.print("u_hat (original)", numClasses*dim);
    v.print("v (original)", numClasses*dim);
    CUUnifiedBlob::vectorVectorScalarProduct(u_hat, v, b, numClasses, flattenedTensorSize, dim);
    CUUnifiedBlob::CUDA_vectorVectorScalarProduct(u_hat, v, b_cuda_output, numClasses, flattenedTensorSize, dim);
    sleep(1);
    b.print("b output (seq)", numClasses);
    b_cuda_output.print("b output (cuda)", numClasses);
}

int main() {
//    test_SingleLayerCNN();
//    test_CapsuleNetSquishing();
//    test_VectorMapFromFeatureMaps
//    test_FeatureMapsFromVectorMap();

//    test_CapsuleNetwork_ForwardPropagation();
//    test_CapsuleNetwork_BackPropagation();
//    test_CapsuleNetwork_getMarginLoss();
//    test_CapsuleNetwork_reconstruction();
//    test_CapsuleNetwork_multipleReconstruction();

//    test_CapsuleNetwork_Epoch();
//    test_NetworkTallyingTiming();

//    test_CUUnifiedBlob_CUDA_matrixVectorMultiplication();
//    test_CUUnifiedBlob_CUDA_softmax();
//    test_CUUnifiedBlob_CUDA_weightReduceAndSquash();
//    test_CUUnifiedBlob_CUDA_vectorSquash();
    test_CUUnifiedBlob_CUDA_getScalarProducts();

//    ConvolutionalNetwork cnn;
//    cnn.init();
//    cnn.train();

//    MultilayerPerceptron mp(784, 10, {10});
//    mp.init();
//    mp.train();
//    mp.tally(false);

    // TODO have all Networks derive from a master 'Network' class

    return 0;
}