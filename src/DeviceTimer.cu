//
// Created by daniellopez on 5/1/18.
//

#include <hip/hip_runtime_api.h>
#include "DeviceTimer.h"

DeviceTimer::DeviceTimer() {
    hipEventCreate(&beginEvent);
    hipEventCreate(&endEvent);
}

DeviceTimer::~DeviceTimer() {
    hipEventDestroy(beginEvent);
    hipEventDestroy(endEvent);
}

void DeviceTimer::start() {
    hipEventRecord(beginEvent);
//    hipEventSynchronize(beginEvent);
    timerOn = true;
}

void DeviceTimer::stop() {
    hipEventRecord(endEvent);
    timerOn = false;
//    hipEventSynchronize(endEvent);
}

long double DeviceTimer::getElapsedTime() const {
    hipEventSynchronize(endEvent);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, beginEvent, endEvent);
    return elapsedTime;
}