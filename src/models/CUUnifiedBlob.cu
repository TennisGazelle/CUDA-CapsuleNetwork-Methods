#include "hip/hip_runtime.h"
//
// Created by daniellopez on 4/4/18.
//

#include <cassert>
#include <CUDAUtils.h>
#include <iostream>
#include <cmath>
#include "models/CUUnifiedBlob.h"
#include "CUDAUtils.h"

using namespace std;

CUUnifiedBlob::CUUnifiedBlob(int pSize) : size(pSize), data(nullptr), isGPUAllocated(false) {
    assert (pSize > 0);
    allocateMemory();
}

CUUnifiedBlob::~CUUnifiedBlob() {
    if (isGPUAllocated) {
        deallocateMemory();
    }
}

void CUUnifiedBlob::allocateMemory() {
    assert(!isGPUAllocated);
    auto error = hipMallocManaged((void**)&data, size * sizeof(double), hipMemAttachGlobal);
    CUDAUtils::handleError(error);
    isGPUAllocated = true;
}

void CUUnifiedBlob::deallocateMemory() {
    assert(isGPUAllocated);
    hipFree(data);
    data = nullptr;
    isGPUAllocated = false;
}

void CUUnifiedBlob::clear() {
    for (int i = 0; i < size; i++) {
        data[i] = 0.0;
    }
}

void CUUnifiedBlob::resize(int newSize) {
    deallocateMemory();
    size = newSize;
    allocateMemory();
}

void CUUnifiedBlob::print(const string& msg, int width) {
    if (!msg.empty()) {
        cout << msg << endl;
    }
    int bufferSize = min(size, 200);
    for (int i = 0; i < bufferSize; i++) {
        cout << data[i] << "\t";
        if (((i+1) % width) == 0) {
            cout << endl;
        }
    }
    cout << endl;
}

bool CUUnifiedBlob::operator==(const CUUnifiedBlob &other) const {
    if (this == &other) {
        return true;
    }

    if (size != other.size) {
        cout << "bad sizes" << endl;
        return false;
    }

    for (int i = 0; i < size; i++) {
        if (data[i] != other.data[i]) {
            cout << "they didn't match at: " << i << endl;
            cout << "this: " << data[i] << " other: " << other.data[i] << endl;
            return false;
        }
    }

    return true;
}

void CUUnifiedBlob::setValueAt_1D(int location, double incomingValue) {
    assert(0 <= location && location < size);
    data[location] = incomingValue;
}

void CUUnifiedBlob::setValueAt_2D(int x, int y, int xDim, double incomingValue) {
    // where is the location?
    int location = x * xDim;
    location += y;

    setValueAt_1D(location, incomingValue);
}

void CUUnifiedBlob::setValueAt_3D(int x, int y, int z, int xDim, int yDim, double incomingValue) {
    int location = z * xDim * yDim;
    location += x * xDim;
    location += y;

    setValueAt_1D(location, incomingValue);
}

void CUUnifiedBlob::matrixVectorMultiplication(CUUnifiedBlob &matrix, CUUnifiedBlob &inputVector,
                                               CUUnifiedBlob &outputVector, int inputDim, int outputDim) {
    assert(matrix.size == inputDim*outputDim);
    assert(inputVector.size == inputDim);
    assert(outputVector.size == outputDim);

    for (int i = 0; i < outputDim; i++) {
        for (int j = 0; j < inputDim; j++) {
            outputVector.data[i] += inputVector.data[j] * matrix.data[i*inputDim + j];
        }
    }
}

void CUUnifiedBlob::vectorVectorSoftmax(CUUnifiedBlob& b, CUUnifiedBlob& c,
                                        int numClasses, int tensorSize) {
    for (int k = 0; k < numClasses; k++) {
        double sum_b_exps = 0.0;
        for (int t = 0; t < tensorSize; t++) {
            sum_b_exps += exp(b.data[t*numClasses + k]);
        }

        // then go through the c's and set accordingly
        for (int t = 0; t < tensorSize; t++) {
            c.data[t*numClasses + k] = exp(b.data[t*numClasses + k])/ sum_b_exps;
        }
    }
}

void CUUnifiedBlob::weightReduceVectors(CUUnifiedBlob &u_hat, CUUnifiedBlob &c, CUUnifiedBlob &v, int numClasses,
                                        int tensorSize, int dim) {
    for (int k = 0; k < numClasses; k++) {
        for (int t = 0; t < tensorSize; t++) {
            int u_hat_index = t*numClasses*dim + k*dim;

            for (int i = u_hat_index; i < u_hat_index + dim; i++) {
                v.data[i % (numClasses*dim)] += u_hat.data[i] * c.data[t*numClasses+k];
            }
        }
    }
}

void CUUnifiedBlob::vectorSquash(CUUnifiedBlob &v, int numVecs, int vecDim) {
    for (int v_index = 0; v_index < numVecs*vecDim; v_index += vecDim) {
        double sum_squares = 0;
    	for (int i = 0; i < vecDim; i++) {
    	    sum_squares += pow(v.data[v_index + i], 2);
    	}
    	double squashFactor = sum_squares / (1.0 + sum_squares);
    	sum_squares = sqrt(sum_squares);
    	for (int i = 0; i < vecDim; i++) {
    		v.data[v_index + i] *= squashFactor / sum_squares;
    	}
    }
}

void CUUnifiedBlob::vectorVectorScalarProduct(CUUnifiedBlob &u_hat, CUUnifiedBlob &v, CUUnifiedBlob &b, int numClasses, int tensorSize, int dim) {
    for (int k = 0; k < numClasses; k++) {
        int v_index = k*dim;
    	for (int t = 0; t < tensorSize; t++) {
    		int u_hat_index = t*numClasses*dim + k*dim;
    		int b_index = t*numClasses + k;

    		for (int i = 0; i < dim; i++) {
    			b.data[b_index] += u_hat.data[u_hat_index + i] * v.data[v_index + i];
    		}
    	}
    }
}

void CUUnifiedBlob::CUDA_matrixVectorMultiplication(CUUnifiedBlob &matrix,
                                                    CUUnifiedBlob &inputVector,
                                                    CUUnifiedBlob &outputVector,
                                                    int inputDim,
                                                    int outputDim,
                                                    int numMultiplications) {
    cu_matrixVectorMultiplication_kernel<<<numMultiplications, outputDim>>>(matrix.data,
                                                    inputVector.data,
                                                    outputVector.data,
                                                    inputDim,
                                                    outputDim);
}

void CUUnifiedBlob::CUDA_vectorVectorSoftmax(CUUnifiedBlob &b,
                                             CUUnifiedBlob &c,
                                             int numClasses,
                                             int tensorSize) {
    int offset = 0;
    int numThreads = min(1024, tensorSize);
    cu_vectorVectorSoftmax_kernel<<<numClasses, numThreads, numThreads*sizeof(double)>>>(b.data, c.data, numClasses, tensorSize);
}

void CUUnifiedBlob::CUDA_weightReduceVectors(CUUnifiedBlob &u_hat,
                                             CUUnifiedBlob &c,
                                             CUUnifiedBlob &v,
                                             int numClasses,
                                             int tensorSize,
                                             int dim) {
    dim3 blockDimensions(numClasses, dim);
    int numThreads = min(1024, tensorSize);
    cu_weightReduceVector_kernel<<<blockDimensions, numThreads, numThreads*sizeof(double)>>>(u_hat.data, c.data, v.data, numClasses, tensorSize, dim);
}

void CUUnifiedBlob::CUDA_vectorSquash(CUUnifiedBlob &v, int numVecs, int vecDim) {
    cu_vectorSquash_kernel<<<numVecs, vecDim, vecDim*sizeof(double)>>>(v.data, numVecs, vecDim);
}

void CUUnifiedBlob::CUDA_vectorVectorScalarProduct(CUUnifiedBlob &u_hat, CUUnifiedBlob &v, CUUnifiedBlob &b, int numClasses, int tensorSize, int dim) {
    dim3 blockDims(tensorSize, numClasses);
    cu_vectorVectorScalarProduct_kernel<<<blockDims, dim, dim*sizeof(double)>>>(u_hat.data, v.data, b.data, numClasses, tensorSize, dim);
}

__global__
void cu_matrixVectorMultiplication_kernel(double *matrix, double *inputVector, double *outputVector,
                                          int inputDim, int outputDim) {
    int u_hat_index = threadIdx.x + (blockIdx.x * outputDim);
    double cache = 0.0;
    for (int c = 0; c < inputDim; c++) {
        cache += matrix[u_hat_index*inputDim+c] * inputVector[blockIdx.x*inputDim+c];
    }
    outputVector[u_hat_index] = cache;
}

__global__
void cu_vectorVectorSoftmax_kernel(double *b, double *c, int numClasses, int tensorSize) {
    int t = threadIdx.x;
    int k = blockIdx.x;

    extern __shared__
    double shared_b_exps[];

    double my_exp_bs [8]; // make this dynamic and only as needed
    for (int i = 0; i*1024 < tensorSize; i++) {
        if (i*1024 + t < tensorSize) {
            my_exp_bs[i] = exp(b[(i*1024+t)*numClasses+k]); // consider using hexp() for speed
            shared_b_exps[t] += my_exp_bs[i];
        }
    }
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (t % (2*s) == 0) {
            shared_b_exps[t] += shared_b_exps[t + s];
        }
        __syncthreads();
    }

    double sum_exps = shared_b_exps[0];
    for (int i = 0; i*1024 < tensorSize; i++) {
        if (i*1024 + t < tensorSize) {
            c[(i*1024+t)*numClasses+k] = my_exp_bs[i] / sum_exps;
        }
    }
}

__global__
void cu_weightReduceVector_kernel(double *u_hat, double *c, double *v, int numClasses, int tensorSize, int dim) {
    int k = blockIdx.x;
    int specificDim = blockIdx.y;
    int t = threadIdx.x;

    int u_hat_index = t*numClasses*dim + k*dim;
    int c_index = t*numClasses+k;
    extern __shared__
    double shared_v_vec[];

    shared_v_vec[t] = u_hat[u_hat_index + specificDim] * c[c_index];
    // if tensorsize > 1024, add them to the shared mem as well
    for (int i = 1; i*1024 < tensorSize; i++) {
        int u_hat_offset = 1024*numClasses*dim;
        int c_offset = 1024*numClasses;
        if (i*1024 + t < tensorSize) {
            shared_v_vec[t] += u_hat[u_hat_index + specificDim + u_hat_offset] * c[c_index + c_offset];
        }
    }
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (t % (2*s) == 0) {
            shared_v_vec[t] += shared_v_vec[t + s];
        }
        __syncthreads();
    }

    if (t == 0) {
        v[k*dim + specificDim] = shared_v_vec[0];
    }
}

__global__
void cu_vectorSquash_kernel(double *v, int numVecs, int vecDim) {
    int v_index = blockIdx.y*gridDim.x + blockIdx.x;
    int v_val_index = threadIdx.x;

    extern __shared__
    double shared_v_values[];
    // reduce the square of the individual elements in shared mem
    if (v_index < numVecs) {
    	shared_v_values[v_val_index] = pow(v[v_index*vecDim + v_val_index], 2);
    }
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (v_val_index % (2*s) == 0) {
        	shared_v_values[v_val_index] += shared_v_values[v_val_index + s];
        }
        __syncthreads();
    }

    // calc squashing func
    if (v_val_index == 0) {
        shared_v_values[1] = shared_v_values[0] / (1 + shared_v_values[0]);
        shared_v_values[0] = sqrt(shared_v_values[0]);
    }
    __syncthreads();

    if (v_index < numVecs) {
        v[v_index*vecDim + v_val_index] *= shared_v_values[1] / shared_v_values[0];
    }
}

__global__
void cu_vectorVectorScalarProduct_kernel(double *u_hat, double *v, double *b, int numClasses, int tensorSize, int dim) {
    int k = blockIdx.y;
    int specificDim = threadIdx.x;
    int t = blockIdx.x;
    int u_hat_index = t*numClasses*dim + k*dim;

    extern __shared__
    double shared_scalar_products[];
    shared_scalar_products[specificDim] = u_hat[u_hat_index + specificDim] * v[k*dim + specificDim];
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s*= 2) {
    	if (specificDim % (2*s) == 0) {
    		shared_scalar_products[specificDim] += shared_scalar_products[specificDim + s];
    	}
    	__syncthreads();
    }

    b[t*numClasses+k] = shared_scalar_products[0];
}