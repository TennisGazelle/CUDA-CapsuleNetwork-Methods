#include "hip/hip_runtime.h"
//
// Created by daniellopez on 4/4/18.
//

#include <cassert>
#include <CUDAUtils.h>
#include <iostream>
#include <cmath>
#include "models/CUUnifiedBlob.h"
#include "CUDAUtils.h"

using namespace std;

CUUnifiedBlob::CUUnifiedBlob(int pSize) : size(pSize), data(nullptr), isGPUAllocated(false) {
    assert (pSize > 0);
    allocateMemory();
}

CUUnifiedBlob::~CUUnifiedBlob() {
    if (isGPUAllocated) {
        deallocateMemory();
    }
}

void CUUnifiedBlob::allocateMemory() {
    assert(!isGPUAllocated);
    auto error = hipMallocManaged((void**)&data, size * sizeof(double), hipMemAttachGlobal);
    CUDAUtils::handleError(error);
    isGPUAllocated = true;
}

void CUUnifiedBlob::deallocateMemory() {
    assert(isGPUAllocated);
    hipFree(data);
    data = nullptr;
    isGPUAllocated = false;
}

void CUUnifiedBlob::clear() {
    for (int i = 0; i < size; i++) {
        data[i] = 0.0;
    }
}

void CUUnifiedBlob::resize(int newSize) {
    deallocateMemory();
    size = newSize;
    allocateMemory();
}

void CUUnifiedBlob::print(const string& msg, int width) {
    if (!msg.empty()) {
        cout << msg << endl;
    }
    int bufferSize = min(size, 1000);
    for (int i = 0; i < bufferSize; i++) {
        cout << data[i] << "\t";
        if (((i+1) % width) == 0) {
            cout << endl;
        }
    }
    cout << endl;
}

void CUUnifiedBlob::setValueAt_1D(int location, double incomingValue) {
    assert(0 <= location && location < size);
    data[location] = incomingValue;
}

void CUUnifiedBlob::setValueAt_2D(int x, int y, int xDim, double incomingValue) {
    // where is the location?
    int location = x * xDim;
    location += y;

    setValueAt_1D(location, incomingValue);
}

void CUUnifiedBlob::setValueAt_3D(int x, int y, int z, int xDim, int yDim, double incomingValue) {
    int location = z * xDim * yDim;
    location += x * xDim;
    location += y;

    setValueAt_1D(location, incomingValue);
}

void CUUnifiedBlob::matrixVectorMultiplication(CUUnifiedBlob &matrix, CUUnifiedBlob &inputVector,
                                               CUUnifiedBlob &outputVector, int inputDim, int outputDim) {
    assert(matrix.size == inputDim*outputDim);
    assert(inputVector.size == inputDim);
    assert(outputVector.size == outputDim);

    for (int i = 0; i < outputDim; i++) {
        for (int j = 0; j < inputDim; j++) {
            cout << "output at += : " << i << " matrix:[" << i  << ", " << j << "]" << endl;
            outputVector.data[i] += inputVector.data[j] * matrix.data[i*inputDim + j];
        }
    }
}

void CUUnifiedBlob::vectorVectorSoftmax(CUUnifiedBlob& b, CUUnifiedBlob& c,
                                        int numClasses, int tensorSize) {
    for (int k = 0; k < numClasses; k++) {
        double sum_b_exps = 0.0;
        for (int t = 0; t < tensorSize; t++) {
            sum_b_exps += exp(b.data[t*numClasses + k]);
        }

        // then go through the c's and set accordingly
        for (int t = 0; t < tensorSize; t++) {
            c.data[t*numClasses + k] = exp(b.data[t*numClasses + k])/ sum_b_exps;
        }
    }
}

void CUUnifiedBlob::CUDA_matrixVectorMultiplication(CUUnifiedBlob &matrix,
                                                    CUUnifiedBlob &inputVector,
                                                    CUUnifiedBlob &outputVector,
                                                    int inputDim,
                                                    int outputDim,
                                                    int numMultiplications) {
    cu_matrixVectorMultiplication_kernel<<<numMultiplications, outputDim>>>(matrix.data,
                                                    inputVector.data,
                                                    outputVector.data,
                                                    inputDim,
                                                    outputDim);
}

void CUUnifiedBlob::CUDA_vectorVectorSoftmax(CUUnifiedBlob &b,
                                             CUUnifiedBlob &c,
                                             int numClasses,
                                             int tensorSize) {
    int offset = 0;
    do {
        unsigned int numThreadsToAllocate = (unsigned int) min(1024, tensorSize);
        cu_vectorVectorSoftmax_kernel<<<numClasses, numThreadsToAllocate, numThreadsToAllocate*sizeof(double)>>>(b.data, c.data, numClasses, tensorSize, offset);
        tensorSize -= numThreadsToAllocate;
        offset++;
    } while (tensorSize > 0);
}

__global__
void cu_matrixVectorMultiplication_kernel(double *matrix, double *inputVector, double *outputVector,
                                          int inputDim, int outputDim) {
    int u_hat_index = threadIdx.x + (blockIdx.x * outputDim);
    double cache = 0.0;
    for (int c = 0; c < inputDim; c++) {
        cache += matrix[u_hat_index*inputDim+c] * inputVector[blockIdx.x*inputDim+c];
    }
    outputVector[u_hat_index] = cache;
}

__global__
void cu_vectorVectorSoftmax_kernel(double *b, double *c, int numClasses, int tensorSize, int offset) {
    int t = threadIdx.x + offset;
    int k = blockIdx.x;

    extern __shared__
    double shared_b_exps[];

    double my_exp_b = exp(b[t*numClasses+k]); // consider using hexp() for speed
    shared_b_exps[t] = my_exp_b;
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (t % (2*s) == 0) {
            shared_b_exps[t] += shared_b_exps[t + s];
        }
        __syncthreads();
    }

    double sum_exps = shared_b_exps[0];
    c[t*numClasses + k] = my_exp_b / sum_exps;
}